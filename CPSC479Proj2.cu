
//Pia Wetzel



/*
Program uses the KNN (with K = 3) classification algorithm to classify the Iris species Setosa, Virginica, and Versicolor. 
Goal is it to identify an Iris species based on the four parameters Sepal-width, Sepal-length, Petal-width, and Petal-length.
*/



#include <stdio.h>
#include <hip/hip_runtime.h>
#include "math.h"

#include<stdio.h>
#include<string.h>



__global__ void knn (double *oMatrix, double *topN, double*knns, unsigned matrixsize) {


  //Extracts the species of the "k" best euclidean distances
  unsigned id = blockIdx.x * blockDim.x + threadIdx.x;

	for(unsigned i = 0 ; i < matrixsize-1; i++)
	{
	 if(oMatrix[i] == topN[id]){
	 knns[id] = oMatrix[i+1];
	}
	}

}

//Calculated Euclidean Distance between each matrix row and a given test vector. 
//The "result" is a matrix with x rows and 2 colums, containing the euclidean 
//distance of the row plus the numerical idenifier of the Iris species associated with the row
__global__ void eucl_dist ( double *matrix, double *test, double *result, double *result2, unsigned matrixsize) {

  unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned max = 4;

  for (unsigned jj = 0; jj < max; ++jj) {
		result[id*2] += (matrix[id * matrixsize+jj] - test[jj])*(matrix[id * matrixsize +jj] - test[jj]);
		result2[id] += (matrix[id * matrixsize+jj] - test[jj])*(matrix[id * matrixsize +jj] - test[jj]);

	if(jj == 0)
	{
		result[id*2 + 1] = matrix[id * matrixsize + 4];
	}
	if(jj == max-1)
	{
		result[id*2] = sqrt(result[2*id]);
		result2[id] = sqrt(result2[id]);
	}

  }
}


//Parallel sorts the matrix with x rows and 2 column, ordered by increasing euclidean distance
__global__ void even_sort(double *arr, unsigned matrixsize) {

	double temp;
	int id = (threadIdx.x)*2;

	if(id <= matrixsize-2)
	{
	if(arr[id] > arr[id+1])
	{
		temp = arr[id];
		arr[id] = arr[id+1];
		arr[id+1] = temp;
	}
	}
}

__global__ void odd_sort(double *arr, unsigned matrixsize) {
	double temp;
	int id = (threadIdx.x)*2+1;

	if(id <= matrixsize-2)
	{
	if(arr[id] > arr[id+1])
	{
		temp = arr[id];
		arr[id] = arr[id+1];
		arr[id+1] = temp;
	}
	}
}



	#define N 147
	#define M 5
int main() {


	const unsigned KNN = 3;

	//Some test values ("randomly" taken out of original data set)

	//double test_iris[5] = {5.6,3.0,4.5,1.5,200};  //I'm a Versicolor
	double test_iris[5] = {5.1,3.5,1.4,0.2, 100}; //I'm a Setosa
	//double test_iris[5] = {6.7,3.0,5.2,2.3,300}; //I'm a Virginica

	dim3 block(N, M, 1);

	double *eucl_distance,*eucl_distance2, *result,*result2, *test, *matrix, *knns, *knnres;

	hipMalloc(&result,2*N*sizeof(double));
	hipMalloc(&result2,N*sizeof(double));

	hipMalloc(&test,(M)*sizeof(double));
	hipMalloc(&matrix,N*M*sizeof(double));
	hipMalloc(&knnres,KNN*sizeof(double));

	eucl_distance = (double *)malloc(2*N * sizeof(double));
	eucl_distance2 = (double *)malloc(N * sizeof(double));
	knns = (double *)malloc(KNN * sizeof(double));


	  //Training data

	//Setosa = 100
	//Versicolor = 200
	//Virginica = 300

	//Data is taken from https://archive.ics.uci.edu/ml/datasets/iris
	double iris2[147][5] ={
	{4.9,3.0,1.4,0.2,100},
	{4.7,3.2,1.3,0.2,100},
	{4.6,3.1,1.5,0.2,100},
	{5.0,3.6,1.4,0.2,100},
	{5.4,3.9,1.7,0.4,100},
	{4.6,3.4,1.4,0.3,100},
	{5.0,3.4,1.5,0.2,100},
	{4.4,2.9,1.4,0.2,100},
	{4.9,3.1,1.5,0.1,100},
	{5.4,3.7,1.5,0.2,100},
	{4.8,3.4,1.6,0.2,100},
	{4.8,3.0,1.4,0.1,100},
	{4.3,3.0,1.1,0.1,100},
	{5.8,4.0,1.2,0.2,100},
	{5.7,4.4,1.5,0.4,100},
	{5.4,3.9,1.3,0.4,100},
	{5.1,3.5,1.4,0.3,100},
	{5.7,3.8,1.7,0.3,100},
	{5.1,3.8,1.5,0.3,100},
	{5.4,3.4,1.7,0.2,100},
	{5.1,3.7,1.5,0.4,100},
	{4.6,3.6,1.0,0.2,100},
	{5.1,3.3,1.7,0.5,100},
	{4.8,3.4,1.9,0.2,100},
	{5.0,3.0,1.6,0.2,100},
	{5.0,3.4,1.6,0.4,100},
	{5.2,3.5,1.5,0.2,100},
	{5.2,3.4,1.4,0.2,100},
	{4.7,3.2,1.6,0.2,100},
	{4.8,3.1,1.6,0.2,100},
	{5.4,3.4,1.5,0.4,100},
	{5.2,4.1,1.5,0.1,100},
	{5.5,4.2,1.4,0.2,100},
	{4.9,3.1,1.5,0.1,100},
	{5.0,3.2,1.2,0.2,100},
	{5.5,3.5,1.3,0.2,100},
	{4.9,3.1,1.5,0.1,100},
	{4.4,3.0,1.3,0.2,100},
	{5.1,3.4,1.5,0.2,100},
	{5.0,3.5,1.3,0.3,100},
	{4.5,2.3,1.3,0.3,100},
	{4.4,3.2,1.3,0.2,100},
	{5.0,3.5,1.6,0.6,100},
	{5.1,3.8,1.9,0.4,100},
	{4.8,3.0,1.4,0.3,100},
	{5.1,3.8,1.6,0.2,100},
	{4.6,3.2,1.4,0.2,100},
	{5.3,3.7,1.5,0.2,100},
	{5.0,3.3,1.4,0.2,100},
	{7.0,3.2,4.7,1.4,200},
	{6.4,3.2,4.5,1.5,200},
	{6.9,3.1,4.9,1.5,200},
	{5.5,2.3,4.0,1.3,200},
	{6.5,2.8,4.6,1.5,200},
	{5.7,2.8,4.5,1.3,200},
	{6.3,3.3,4.7,1.6,200},
	{4.9,2.4,3.3,1.0,200},
	{6.6,2.9,4.6,1.3,200},
	{5.2,2.7,3.9,1.4,200},
	{5.0,2.0,3.5,1.0,200},
	{5.9,3.0,4.2,1.5,200},
	{6.0,2.2,4.0,1.0,200},
	{6.1,2.9,4.7,1.4,200},
	{5.6,2.9,3.6,1.3,200},
	{6.7,3.1,4.4,1.4,200},
	{5.8,2.7,4.1,1.0,200},
	{6.2,2.2,4.5,1.5,200},
	{5.6,2.5,3.9,1.1,200},
	{5.9,3.2,4.8,1.8,200},
	{6.1,2.8,4.0,1.3,200},
	{6.3,2.5,4.9,1.5,200},
	{6.1,2.8,4.7,1.2,200},
	{6.4,2.9,4.3,1.3,200},
	{6.6,3.0,4.4,1.4,200},
	{6.8,2.8,4.8,1.4,200},
	{6.7,3.0,5.0,1.7,200},
	{6.0,2.9,4.5,1.5,200},
	{5.7,2.6,3.5,1.0,200},
	{5.5,2.4,3.8,1.1,200},
	{5.5,2.4,3.7,1.0,200},
	{5.8,2.7,3.9,1.2,200},
	{6.0,2.7,5.1,1.6,200},
	{5.4,3.0,4.5,1.5,200},
	{6.0,3.4,4.5,1.6,200},
	{6.7,3.1,4.7,1.5,200},
	{6.3,2.3,4.4,1.3,200},
	{5.6,3.0,4.1,1.3,200},
	{5.5,2.5,4.0,1.3,200},
	{5.5,2.6,4.4,1.2,200},
	{6.1,3.0,4.6,1.4,200},
	{5.8,2.6,4.0,1.2,200},
	{5.0,2.3,3.3,1.0,200},
	{5.6,2.7,4.2,1.3,200},
	{5.7,3.0,4.2,1.2,200},
	{5.7,2.9,4.2,1.3,200},
	{6.2,2.9,4.3,1.3,200},
	{5.1,2.5,3.0,1.1,200},
	{5.7,2.8,4.1,1.3,200},
	{6.3,3.3,6.0,2.5,300},
	{5.8,2.7,5.1,1.9,300},
	{7.1,3.0,5.9,2.1,300},
	{6.3,2.9,5.6,1.8,300},
	{6.5,3.0,5.8,2.2,300},
	{7.6,3.0,6.6,2.1,300},
	{4.9,2.5,4.5,1.7,300},
	{7.3,2.9,6.3,1.8,300},
	{6.7,2.5,5.8,1.8,300},
	{7.2,3.6,6.1,2.5,300},
	{6.5,3.2,5.1,2.0,300},
	{6.4,2.7,5.3,1.9,300},
	{6.8,3.0,5.5,2.1,300},
	{5.7,2.5,5.0,2.0,300},
	{5.8,2.8,5.1,2.4,300},
	{6.4,3.2,5.3,2.3,300},
	{6.5,3.0,5.5,1.8,300},
	{7.7,3.8,6.7,2.2,300},
	{7.7,2.6,6.9,2.3,300},
	{6.0,2.2,5.0,1.5,300},
	{6.9,3.2,5.7,2.3,300},
	{5.6,2.8,4.9,2.0,300},
	{7.7,2.8,6.7,2.0,300},
	{6.3,2.7,4.9,1.8,300},
	{6.7,3.3,5.7,2.1,300},
	{7.2,3.2,6.0,1.8,300},
	{6.2,2.8,4.8,1.8,300},
	{6.1,3.0,4.9,1.8,300},
	{6.4,2.8,5.6,2.1,300},
	{7.2,3.0,5.8,1.6,300},
	{7.4,2.8,6.1,1.9,300},
	{7.9,3.8,6.4,2.0,300},
	{6.4,2.8,5.6,2.2,300},
	{6.3,2.8,5.1,1.5,300},
	{6.1,2.6,5.6,1.4,300},
	{7.7,3.0,6.1,2.3,300},
	{6.3,3.4,5.6,2.4,300},
	{6.4,3.1,5.5,1.8,300},
	{6.0,3.0,4.8,1.8,300},
	{6.9,3.1,5.4,2.1,300},
	{6.7,3.1,5.6,2.4,300},
	{6.9,3.1,5.1,2.3,300},
	{5.8,2.7,5.1,1.9,300},
	{6.8,3.2,5.9,2.3,300},
	{6.7,3.3,5.7,2.5,300},
	{6.3,2.5,5.0,1.9,300},
	{6.5,3.0,5.2,2.0,300},
	{6.2,3.4,5.4,2.3,300},
	{5.9,3.0,5.1,1.8,300}};




	hipMemcpy(matrix,iris2, N*M*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(test,test_iris, (M)*sizeof(double), hipMemcpyHostToDevice);

	eucl_dist<<<1, N>>>(matrix, test, result,result2, M);
	hipMemcpy(eucl_distance, result, 2*N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(eucl_distance2, result2, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(result2,eucl_distance2, N*sizeof(double), hipMemcpyHostToDevice);
		
		for(unsigned i = 0; i <= N/2; i++){

			even_sort<<<1, N>>>(result2, N);
			odd_sort<<<1, N>>>(result2, N);
		}

	hipMemcpy(eucl_distance2, result2, N*sizeof(double), hipMemcpyDeviceToHost);


	hipMemcpy(result,eucl_distance, 2*N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(result2,eucl_distance2, N*sizeof(double), hipMemcpyHostToDevice);
	knn<<<1,3>>>(result, result2, knnres, 2*N);

	hipMemcpy(knns, knnres, KNN*sizeof(double), hipMemcpyDeviceToHost);
	unsigned versicolor, virginica, setosa;



	   for (unsigned i = 0; i < KNN; ++i) {
		  if(knns[i] == 100){setosa++;}
		  else if(knns[i] == 200){versicolor++;}
		  else if (knns[i] == 300){virginica++;}
	    }

	    printf("\n--------------------------------------------------------\n");
	    printf("\n\n\nInput:\n\nSepal-length: %2f\nSepal-width: %2f\nPetal-length: %2f\nPetal-width: %2f", test_iris[0], test_iris[1], test_iris[2], test_iris[3]);
	    printf("\n\nThe %2d closest neighbors:\nsetosa: %2d virginica: %2d versicolor: %2d",KNN, setosa, virginica, versicolor);
	    printf("\n\nApplying KNN classification with k=%2d yields: ", KNN);

	    if(setosa > virginica && setosa > versicolor)
	    {
	      printf("The input is a Setosa\n\n");
	    }else if(virginica > setosa && virginica > versicolor)
	    {
	      printf("The input is a Virginica\n\n");
	    }else if(versicolor > setosa && versicolor > virginica)
	    {
	      printf("The input is a Versicolor\n\n");
	    }
	    else
	    {
	      printf("There is a tie! Try different values.\n\n");
	    }

	    printf("--------------------------------------------------------\n");
	    if(test_iris[4] == 100){printf("\nCorrect answer: Setosa\n\n");}
	    else if(test_iris[4] == 200){printf("\nCorrect answer: Versicolor\n\n");}
	    else if(test_iris[4] == 300){printf("\nCorrect answer: Virginica\n\n");}
	    return 0;

	}
